#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t evaluarCuda(float *c, float vi, int rl, float vd, size_t size);

__global__ void objetivo(float *c, float vi, int rl, float vd)
{
	// obtener coordenada del espacio de busqueda
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// obtener r1 y r2 en formato R1_ R1_ R1_ R1_ R2_ R2_ R2_ R2_

	int r1 = i / 1000;
	int r2 = i % 1000;

	// getOhms()
	int banda1 = r1 / 100;
	int banda2 = (r1 / 10) % 10;
	int m = r1 % 10;
	int multiplicador = 1;

	for (int k = 0; k < m; k++){
		multiplicador *= 10;
	}

	int ohmsR1 = ((banda1 * 10) + banda2) * multiplicador;

	// getOhms para R2
	banda1 = r2 / 100;
	banda2 = (r2 / 10) % 10;
	m = r2 % 10;
	multiplicador = 1;

	for (int k = 0; k < m; k++){
		multiplicador *= 10;
	}

	int ohmsR2 = ((banda1 * 10) + banda2) * multiplicador;
	
	int req = (ohmsR1*ohmsR2) / (ohmsR1 + ohmsR2);	// resistencia equivalente

	float vl = vi * ((ohmsR1*rl) / (ohmsR1 + rl)) / (ohmsR2 + (ohmsR1*rl) / (ohmsR1 + rl));	// voltaje en la carga

	// minimizar funcion objetivo
	float f = 0.4*(req - rl)*(req - rl) + (vl - vd)*(vl - vd);
	c[i] = f;

	//c[i] = i;
	//printf("thread %d: \n", i);

}



//Rutina 'lanzadora' del "kernel"
hipError_t evaluarCuda(float *c, float vi, int rl, float vd, size_t size)
{
	//float *dev_a = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;

	//Seleccionar el GPU a utilizar
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("Error: hipSetDevice\n");
		goto Error;
	}

	//Solicitar memoria para los vectores a utilizar dentro del GPU
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("Error: hipMalloc\n");
		goto Error;
	}
	/*
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("Error: hipMalloc\n");
		goto Error;
	}
	


	//Copiar los datos a la memoria del GPU
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("Error: hipMalloc\n");
		goto Error;
	}
	*/

	int BLOCK_SIZE, grid;

	if (size < 512){
		BLOCK_SIZE = size;
		grid = 1;
	}
	else
	{
		BLOCK_SIZE = 512;
		grid = size / BLOCK_SIZE;
	}

	//dim3 threadBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 grid(K, K);

	//'Lanzar' el "kernel"
	objetivo << <grid, BLOCK_SIZE >> >(dev_c, vi, rl, vd);

	//Esperar a que terminen los "threads"
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("Error: hipDeviceSynchronize (c�digo de error: %d reportado por objetivo)\n", cudaStatus);
		goto Error;
	}

	//Recuperar los resultados de la memoria del GPU
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("Error: hipMemcpy\n");
		goto Error;
	}

Error:
	hipFree(dev_c);
	//hipFree(dev_a);

	return cudaStatus;
}
